
#include <hip/hip_runtime.h>
#include<iostream>
#include<iomanip>
#include<array>
template<typename T>
class interval{
public:
    T x,y;
    __device__ interval<T> sqrt(){
        return interval<T>{__dsqrt_rd(x),__dsqrt_ru(y)};
    }
};

template<typename T>
__device__ interval<T> operator+(const interval<T>& left, const interval<T>& right){
    return interval<T>{0,0};
}
template<>
__device__ interval<double> operator+(const interval<double>& left, const interval<double>& right){
    return interval<double>{__dadd_rd(left.x,right.x),__dadd_ru(left.y,right.y)};
}

template<typename T>
__device__ interval<T> operator-(const interval<T>& left, const interval<T>& right){
    return interval<T>{0,0};
}
template<>
__device__ interval<double> operator-(const interval<double>& left, const interval<double>& right){
    return interval<double>{__dsub_rd(left.x,right.x),__dsub_ru(left.y,right.y)};
}

template<typename T>
__device__ interval<T> operator*(const interval<T>& left, const interval<T>& right){
    return interval<T>{0,0};
}
template<>
__device__ interval<double> operator*(const interval<double>& left, const interval<double>& right){
    if(right.y<0){
        if(left.y<0){
            return interval<double>{__dmul_rd(left.y,right.y),__dmul_ru(left.x,right.x)};
        }else if(left.x>0){
            return interval<double>{__dmul_rd(left.y,right.x),__dmul_ru(left.x,right.y)};
        }else return interval<double>{__dmul_rd(left.y,right.x),__dmul_ru(left.x,right.x)};
    }else if(right.x>0){
        if(left.y<0){
            return interval<double>{__dmul_rd(left.x,right.y),__dmul_ru(left.y,right.x)};
        }else if(left.x>0){
            return interval<double>{__dmul_rd(left.x,right.x),__dmul_ru(left.y,right.y)};
        }else return interval<double>{__dmul_rd(left.x,right.y),__dmul_ru(left.y,right.y)};
    }else{
        if(left.y<0){
            return interval<double>{__dmul_rd(left.x,right.y),__dmul_ru(left.x,right.x)};
        }else if(left.x>0){
            return interval<double>{__dmul_rd(left.y,right.x),__dmul_ru(left.y,right.y)};
        }else {
            double x1=__dmul_rd(left.x,right.y);
            double x2=__dmul_rd(left.y,right.x);
            double y1=__dmul_ru(left.x,right.x);
            double y2=__dmul_ru(left.y,right.y);
            return interval<double>{x1<x2?x1:x2,y1>y2?y1:y2};
        }
    }
}

template<typename T>
__device__ interval<T> operator/(const interval<T>& left, const interval<T>& right){
    return interval<T>{0,0};
}
template<>
__device__ interval<double> operator/(const interval<double>& left, const interval<double>& right){
    if(right.y<0){
        if(left.y<0){
            return interval<double>{__ddiv_rd(left.y,right.x),__ddiv_ru(left.x,right.y)};
        }else if(left.x>0){
            return interval<double>{__ddiv_rd(left.y,right.y),__ddiv_ru(left.x,right.x)};
        }else return interval<double>{__ddiv_rd(left.y,right.y),__ddiv_ru(left.x,right.y)};
    }else{
        if(left.y<0){
            return interval<double>{__ddiv_rd(left.x,right.x),__ddiv_ru(left.y,right.y)};
        }else if(left.x>0){
            return interval<double>{__ddiv_rd(left.x,right.y),__ddiv_ru(left.y,right.x)};
        }else return interval<double>{__ddiv_rd(left.x,right.x),__ddiv_ru(left.y,right.x)};
    }
}

__global__ void kernel(interval<double> a,interval<double> b, interval<double> *c) {
    c[0]=a+b;
    c[1]=a-b;
    c[2]=a*b;
    c[3]=a/b;
    c[4]=a.sqrt();
    }
    
int main() {
    std::array<interval<double>,5> c;
    interval<double> *dev_c;
    hipMalloc(reinterpret_cast<void**>(&dev_c),sizeof(interval<double>)*5);
    
    kernel<<<1,1>>>(interval<double>{0.333333333333333,1},interval<double>{0.999999999999999,3},dev_c);
    hipDeviceSynchronize();
    hipMemcpy(c.data(),dev_c,sizeof(interval<double>)*5,hipMemcpyDeviceToHost);
    hipFree(dev_c);
    
    for(const auto& v:c){
        std::cout<<"["<<std::fixed<<std::setprecision(15)<<v.x<<", "<<v.y<<"]"<<std::endl;
    }
    return 0;
}